#include "hip/hip_runtime.h"
#include "maskGauss.h"

#include <opencv2\cudaimgproc.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h> // Ensure this header is included for CUDA math functions

#include <vector>
#include <iostream>


__global__ void computeWeightedAverageKernel(const uchar* mask, const cv::Point* contourPoints, const float* contourValues,
	float* output, int rows, int cols, int numContourPoints, float sigma2)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < cols && y < rows && mask[y * cols + x] != 0)
	{
		float weightedSum = 0.0;
		float weightTotal = 0.0;

		for (int n = 0; n < numContourPoints; ++n)
		{
			const cv::Point& cp = contourPoints[n];
			const float distance2 = (float(x - cp.x) * float(x - cp.x) + float(y - cp.y) * float(y - cp.y));
			const float weight = exp(-(distance2 / (2.f * sigma2)));
			//const float weight = 1.f / (1 + (distance2 / sigma2));
			const float intensity = contourValues[n];

			weightedSum += intensity * weight;
			weightTotal += weight;
		}

		output[y * cols + x] = (weightTotal > 0) ? weightedSum / weightTotal : 0.f;
	}
}

//bool computeWeightedAverage(const uchar* mask, const cv::Point* contourPoints, const float* contourValues,
//	float* output, int rows, int cols, int numContourPoints, float sigma)
//{
//	uchar* d_mask;
//	cv::Point* d_contourPoints;
//	float* d_contourValues;
//	float* d_output;
//
//	size_t maskSize = rows * cols * sizeof(uchar);
//	size_t pointSize = numContourPoints * sizeof(cv::Point);
//	size_t valueSize = numContourPoints * sizeof(float);
//	size_t outputSize = rows * cols * sizeof(float);
//
//	hipMallocManaged(&d_mask, maskSize);
//	hipMallocManaged(&d_contourPoints, pointSize);
//	hipMallocManaged(&d_contourValues, valueSize);
//	hipMallocManaged(&d_output, outputSize);
//
//	hipMemcpy(d_mask, mask, maskSize, hipMemcpyHostToDevice);
//	hipMemcpy(d_contourPoints, contourPoints, pointSize, hipMemcpyHostToDevice);
//	hipMemcpy(d_contourValues, contourValues, valueSize, hipMemcpyHostToDevice);
//
//	dim3 blockSize(32, 32);
//	dim3 gridSize((cols + blockSize.x - 1) / blockSize.x, (rows + blockSize.y - 1) / blockSize.y);
//
//	// Ensure that gridSize.x and gridSize.y are within valid limits
//	if (gridSize.x > 65535 || gridSize.y > 65535) {
//		std::cout << "Grid size exceeds maximum limit." << std::endl;
//		return false;
//	}
//
//	hipDeviceProp_t prop;
//	hipGetDeviceProperties(&prop, 0);
//	if (blockSize.x * blockSize.y * blockSize.z > (size_t)prop.maxThreadsPerBlock)
//	{
//		std::cout << "Max threads per block: " << prop.maxThreadsPerBlock << "\n";
//		hipFree(d_mask);
//		hipFree(d_contourPoints);
//		hipFree(d_contourValues);
//		hipFree(d_output);
//		return false;
//	}
//
//	computeWeightedAverageKernel << <gridSize, blockSize >> > (d_mask, d_contourPoints, d_contourValues, d_output,
//		rows, cols, numContourPoints, sigma*sigma);
//
//	hipError_t e = hipGetLastError();
//	if (e != hipSuccess)
//	{
//		std::cout << "CUDA Launch Error: " << hipGetErrorString(e) << "\n";
//		hipFree(d_mask);
//		hipFree(d_contourPoints);
//		hipFree(d_contourValues);
//		hipFree(d_output);
//		return false;
//	}
//
//
//	e = hipDeviceSynchronize();
//	if (e != hipSuccess)
//	{
//		std::cout << "CUDA Sync Error: " << hipGetErrorString(e) << "\n";
//		hipFree(d_mask);
//		hipFree(d_contourPoints);
//		hipFree(d_contourValues);
//		hipFree(d_output);
//		return false;
//	}
//
//
//	hipMemcpy(output, d_output, outputSize, hipMemcpyDeviceToHost);
//
//	hipFree(d_mask);
//	hipFree(d_contourPoints);
//	hipFree(d_contourValues);
//	hipFree(d_output);
//	return true;
//}

cv::Mat maskedGaussianGPU(const cv::Mat& grayscale, const cv::Mat& mask, const cv::Mat& contour, float sigma)
{
	if (grayscale.empty() || mask.empty() || contour.empty()) 
	{
		std::cout << "Could not load images.\n";
		return cv::Mat();
	}

	if (grayscale.type() != CV_32F || mask.type() != CV_8UC1 || contour.type() != CV_8UC1) {
		std::cout << "Unexpected image types.\n";
		return cv::Mat();
	}

	// Extract contour points
	std::vector <cv::Point> contourPoints;
	std::vector<float> contourValues;
	for (int y = 0; y < contour.rows; ++y) {
		for (int x = 0; x < contour.cols; ++x) {
			if (contour.at<uchar>(y, x) > 0) {
				contourPoints.emplace_back(x, y);
				contourValues.emplace_back(grayscale.at<float>(x, y));
			}
		}
	}


	// computeWeightedAverage(mask.ptr<uchar>(0), &contourPoints[0], &contourValues[0], output.ptr<float>(0), grayscale.rows, grayscale.cols, contourPoints.size(), sigma);
	
	const size_t numContourPoints{ contourPoints.size() };

	size_t pointSize = numContourPoints * sizeof(cv::Point);
	size_t valueSize = numContourPoints * sizeof(float);


	cv::Point* d_contourPoints;
	float* d_contourValues;

	hipMallocManaged(&d_contourPoints, pointSize);
	hipMallocManaged(&d_contourValues, valueSize);
	hipMemcpy(d_contourPoints, &contourPoints[0], pointSize, hipMemcpyHostToDevice);
	hipMemcpy(d_contourValues, &contourValues[0], valueSize, hipMemcpyHostToDevice);

	const int rows{ grayscale.rows };
	const int cols{ grayscale.cols };

	dim3 blockSize(32, 32);
	dim3 gridSize((cols + blockSize.x - 1) / blockSize.x, (rows + blockSize.y - 1) / blockSize.y);

	// Ensure that gridSize.x and gridSize.y are within valid limits
	if (gridSize.x > 65535 || gridSize.y > 65535) {
		std::cout << "Grid size exceeds maximum limit." << std::endl;
		return cv::Mat();
	}

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	if (blockSize.x * blockSize.y * blockSize.z > (size_t)prop.maxThreadsPerBlock)
	{
		hipFree(d_contourPoints);
		hipFree(d_contourValues);
		std::cout << "Max threads per block: " << prop.maxThreadsPerBlock << "\n";
		return cv::Mat();
	}
	
	cv::cuda::GpuMat maskGpu(mask);
	cv::cuda::GpuMat outputGpu(grayscale.size(), CV_32F);
	computeWeightedAverageKernel << <gridSize, blockSize >> > (maskGpu.ptr<uchar>(0), d_contourPoints, d_contourValues, outputGpu.ptr<float>(0),
		rows, cols, numContourPoints, sigma * sigma);

	hipError_t e = hipGetLastError();
	if (e != hipSuccess)
	{
		hipFree(d_contourPoints);
		hipFree(d_contourValues);
		std::cout << "CUDA Launch Error: " << hipGetErrorString(e) << "\n";
		return cv::Mat();
	}

	e = hipDeviceSynchronize();
	if (e != hipSuccess)
	{
		hipFree(d_contourPoints);
		hipFree(d_contourValues);
		std::cout << "CUDA Sync Error: " << hipGetErrorString(e) << "\n";
		return cv::Mat();
	}

	hipFree(d_contourPoints);
	hipFree(d_contourValues);

	cv::Mat output;
	outputGpu.download(output);

	return output;
}