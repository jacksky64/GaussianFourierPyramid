#include "hip/hip_runtime.h"
#include "maskGauss.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h> // Ensure this header is included for CUDA math functions
#include <opencv2/core.hpp>

#include <vector>
#include <iostream>

#define cudaCheckError() {                                           \
    hipError_t e = hipGetLastError();                              \
    if (e != hipSuccess) {                                          \
        std::cout << "CUDA Launch Error: " << hipGetErrorString(e) << "\n";    \
    }                                                                \
}

#define cudaSynchronize() {                                           \
    e = hipDeviceSynchronize();                                     \
    if (e != hipSuccess) {                                          \
        std::cout << "CUDA Sync Error: " << hipGetErrorString(e) << "\n";      \
    }                                                                \
}


__global__ void computeWeightedAverageKernel(const uchar* mask, const cv::Point* contourPoints, const float* contourValues,
	float* output, int rows, int cols, int numContourPoints, float sigma2)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < cols && y < rows && mask[y * cols + x] != 0)
	{
		float weightedSum = 0.0;
		float weightTotal = 0.0;

		for (int n = 0; n < numContourPoints; ++n)
		{
			const cv::Point& cp = contourPoints[n];
			const float distance2 = (float(x - cp.x) * float(x - cp.x) + float(y - cp.y) * float(y - cp.y));
			const float weight = exp(-(distance2 / (2.f * sigma2)));
			const float intensity = contourValues[n];

			weightedSum += intensity * weight;
			weightTotal += weight;
		}

		output[y * cols + x] = (weightTotal > 0) ? weightedSum / weightTotal : 0.f;
	}
}

bool computeWeightedAverage(const uchar* mask, const cv::Point* contourPoints, const float* contourValues,
	float* output, int rows, int cols, int numContourPoints, float sigma)
{
	uchar* d_mask;
	cv::Point* d_contourPoints;
	float* d_contourValues;
	float* d_output;

	size_t maskSize = rows * cols * sizeof(uchar);
	size_t pointSize = numContourPoints * sizeof(cv::Point);
	size_t valueSize = numContourPoints * sizeof(float);
	size_t outputSize = rows * cols * sizeof(float);

	hipMallocManaged(&d_mask, maskSize);
	hipMallocManaged(&d_contourPoints, pointSize);
	hipMallocManaged(&d_contourValues, valueSize);
	hipMallocManaged(&d_output, outputSize);

	hipMemcpy(d_mask, mask, maskSize, hipMemcpyHostToDevice);
	hipMemcpy(d_contourPoints, contourPoints, pointSize, hipMemcpyHostToDevice);
	hipMemcpy(d_contourValues, contourValues, valueSize, hipMemcpyHostToDevice);

	dim3 blockSize(32, 32);
	dim3 gridSize((cols + blockSize.x - 1) / blockSize.x, (rows + blockSize.y - 1) / blockSize.y);

	// Ensure that gridSize.x and gridSize.y are within valid limits
	if (gridSize.x > 65535 || gridSize.y > 65535) {
		std::cout << "Grid size exceeds maximum limit." << std::endl;
		return false;
	}

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	if (blockSize.x * blockSize.y * blockSize.z > (size_t)prop.maxThreadsPerBlock)
	{
		std::cout << "Max threads per block: " << prop.maxThreadsPerBlock << "\n";
		hipFree(d_mask);
		hipFree(d_contourPoints);
		hipFree(d_contourValues);
		hipFree(d_output);
		return false;
	}

	computeWeightedAverageKernel << <gridSize, blockSize >> > (d_mask, d_contourPoints, d_contourValues, d_output,
		rows, cols, numContourPoints, sigma*sigma);

	hipError_t e = hipGetLastError();
	if (e != hipSuccess)
	{
		std::cout << "CUDA Launch Error: " << hipGetErrorString(e) << "\n";
		hipFree(d_mask);
		hipFree(d_contourPoints);
		hipFree(d_contourValues);
		hipFree(d_output);
		return false;
	}


	e = hipDeviceSynchronize();
	if (e != hipSuccess)
	{
		std::cout << "CUDA Sync Error: " << hipGetErrorString(e) << "\n";
		hipFree(d_mask);
		hipFree(d_contourPoints);
		hipFree(d_contourValues);
		hipFree(d_output);
		return false;
	}


	hipMemcpy(output, d_output, outputSize, hipMemcpyDeviceToHost);

	hipFree(d_mask);
	hipFree(d_contourPoints);
	hipFree(d_contourValues);
	hipFree(d_output);
	return true;
}

